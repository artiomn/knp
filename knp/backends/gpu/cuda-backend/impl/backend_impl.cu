#include "hip/hip_runtime.h"
/**
 * @file backend.cu
 * @brief CUDABackendImpl backend class implementation.
 * @kaspersky_support Artiom N.
 * @date 24.02.2025
 * @license Apache 2.0
 * @copyright © 2024 AO Kaspersky Lab
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <knp/devices/gpu_cuda.h>
#include <knp/meta/assert_helpers.h>
#include <knp/meta/stringify.h>
#include <knp/meta/variant_helpers.h>

#include <spdlog/spdlog.h>

#include <limits>
#include <vector>

#include <boost/mp11.hpp>

#include "backend_impl.cuh"
#include "projection.cuh"
#include "population.cuh"

#include "cuda_lib/get_blocks_config.cuh"
#include "cuda_lib/vector.cuh"


namespace knp::backends::gpu::cuda
{


/*template <typename AllVariants, typename SupportedVariants>
SupportedVariants convert_variant(const AllVariants &input)
{
    SupportedVariants result = std::visit([](auto &&arg) { return arg; }, input);
    return result;
}*/


template <class ProjectionType>
bool is_forcing()
{
    return false;
}


template <>
bool is_forcing<cuda::CUDAProjection<synapse_traits::DeltaSynapse>>()
{
    return true;
}


__global__ void calculate_populations_kernel(cuda::CUDABackendImpl *backend,
                                             typename cuda::CUDABackendImpl::PopulationContainer &populations,
                                             std::uint64_t step)
{
    // Calculate populations. This is the same as inference.

    for (auto &population : populations)
    {
/*        ::cuda::std::visit(
            //[this](auto &arg)
            [backend](auto &arg)
            {
//                using T = std::decay_t<decltype(arg)>;
//                auto message_opt = backend->calculate_population(arg);
            },
            population);
*/
    }
}


__global__ void calculate_projections_kernel(cuda::CUDABackendImpl *backend,
                                             typename cuda::CUDABackendImpl::ProjectionContainer &projections,
                                             std::uint64_t step)
{
    // Calculate projections.
    for (auto &projection : projections)
    {
/*
        std::visit(
            // [this, &projection](auto &arg)
            [backend](auto &arg)
            {
//                using T = std::decay_t<decltype(arg)>;
//                backend->calculate_projection(arg, projection.messages_);
            },
            projection);
*/
    }
}


void CUDABackendImpl::calculate_populations(std::uint64_t step)
{
    // Calculate populations. This is the same as inference.
    // Calculate projections.
    auto [num_blocks, num_threads] = device_lib::get_blocks_config(device_populations_.size());

    calculate_populations_kernel<<<num_blocks, num_threads>>>(this, device_populations_, step);
    hipDeviceSynchronize();
}


void CUDABackendImpl::calculate_projections(std::uint64_t step)
{
    // Calculate projections.
    auto [num_blocks, num_threads] = device_lib::get_blocks_config(device_projections_.size());

    calculate_projections_kernel<<<num_blocks, num_threads>>>(this, device_projections_, step);
    hipDeviceSynchronize();
}


void CUDABackendImpl::load_populations(const knp::backends::gpu::CUDABackend::PopulationContainer &populations)
{
    SPDLOG_DEBUG("Loading populations [{}]...", populations.size());

    device_populations_.clear();
    device_populations_.reserve(populations.size());

    for (const auto &population : populations)
    {
        ::std::visit(
            [this](auto &arg)
            {
                using CPUPopulationType = std::decay_t<decltype(arg)>;

                auto pop = CUDAPopulation<typename CPUPopulationType::PopulationNeuronType>(arg);
                device_populations_.push_back(pop);
            },
            population);
    }
    SPDLOG_DEBUG("All populations loaded.");
}


void CUDABackendImpl::load_projections(const knp::backends::gpu::CUDABackend::ProjectionContainer &projections)
{
    SPDLOG_DEBUG("Loading projections [{}]...", projections.size());

    device_projections_.clear();
    device_projections_.reserve(projections.size());

    for (const auto &projection : projections)
    {
        ::std::visit(
            [this](auto &arg)
            {
                using CPUProjectionType = std::decay_t<decltype(arg)>;
                auto proj = CUDAProjection<typename CPUProjectionType::ProjectionSynapseType>(arg);

                device_projections_.push_back(proj);
            },
            projection);
    }

    SPDLOG_DEBUG("All projections loaded.");
}


void CUDABackendImpl::_init()
{
//    SPDLOG_DEBUG("Initializing CUDABackendImpl backend...");

    // knp::backends::cpu::init(projections_, get_message_endpoint());
//    SPDLOG_DEBUG("Initialization finished.");
}


__device__ std::optional<knp::backends::gpu::cuda::SpikeMessage> CUDABackendImpl::calculate_population(
    CUDAPopulation<knp::neuron_traits::BLIFATNeuron> &population,
    knp::backends::gpu::cuda::device_lib::CUDAVector<cuda::SynapticImpactMessage> &messages,
    std::uint64_t step_n)
{
    // TODO rework
    for (size_t i = 0; i < population.neurons_.size(); ++i)
    {
        neuron_traits::neuron_parameters<neuron_traits::BLIFATNeuron> neuron = population.neurons_[i];
        ++neuron.n_time_steps_since_last_firing_;

        neuron.dynamic_threshold_ *= neuron.threshold_decay_;
        neuron.postsynaptic_trace_ *= neuron.postsynaptic_trace_decay_;
        neuron.inhibitory_conductance_ *= neuron.inhibitory_conductance_decay_;

        /*
        if constexpr (has_dopamine_plasticity<BlifatLikeNeuron>())
        {
            neuron.dopamine_value_ = 0.0;
            neuron.is_being_forced_ = false;
        }
        */

        if (neuron.bursting_phase_ && !--neuron.bursting_phase_)
        {
            neuron.potential_ = neuron.potential_ * neuron.potential_decay_ + neuron.reflexive_weight_;
        }
        else
        {
            neuron.potential_ *= neuron.potential_decay_;
        }
        neuron.pre_impact_potential_ = neuron.potential_;

        population.neurons_[i] = neuron;
    }

    // process_inputs(population, messages);
    for (const cuda::SynapticImpactMessage &message : messages)
    {
        for (size_t i = 0; i < message.impacts_.size(); ++i)
        {
            const auto &impact = message.impacts_[i];

            neuron_traits::neuron_parameters<neuron_traits::BLIFATNeuron> neuron =
                population.neurons_[impact.postsynaptic_neuron_index_];

            // impact_neuron<BlifatLikeNeuron>(neuron, impact.synapse_type_, impact.impact_value_);
            switch (impact.synapse_type_)
            {
                case knp::synapse_traits::OutputType::EXCITATORY:
                    neuron.potential_ += impact.impact_value_;
                    break;
                case knp::synapse_traits::OutputType::INHIBITORY_CURRENT:
                    neuron.potential_ -= impact.impact_value_;
                    break;
                case knp::synapse_traits::OutputType::INHIBITORY_CONDUCTANCE:
                    neuron.inhibitory_conductance_ += impact.impact_value_;
                    break;
                case knp::synapse_traits::OutputType::DOPAMINE:
                    neuron.dopamine_value_ += impact.impact_value_;
                    break;
                case knp::synapse_traits::OutputType::BLOCKING:
                    neuron.total_blocking_period_ = static_cast<unsigned int>(impact.impact_value_);
                    break;
            }

            /*if constexpr (has_dopamine_plasticity<BlifatLikeNeuron>())
            {
                if (impact.synapse_type_ == synapse_traits::OutputType::EXCITATORY)
                {
                    neuron.is_being_forced_ |= message.is_forcing_;
                }
            }*/
            population.neurons_[impact.postsynaptic_neuron_index_] = neuron;
        }
    }

    device_lib::CUDAVector<uint32_t> neuron_indexes;

    // calculate_neurons_post_input_state(population, neuron_indexes);
    for (size_t index = 0; index < population.neurons_.size(); ++index)
    {
        bool spike = false;
        neuron_traits::neuron_parameters<neuron_traits::BLIFATNeuron> neuron = population.neurons_[index];

        if (neuron.total_blocking_period_ <= 0)
        {
            // TODO: Make it more readable, don't be afraid to use if operators.
            // Restore potential that the neuron had before impacts.
            neuron.potential_ = neuron.pre_impact_potential_;
            bool was_negative = neuron.total_blocking_period_ < 0;
            // If it is negative, increase by 1.
            neuron.total_blocking_period_ += was_negative;
            // If it is now zero, but was negative before, increase it to max, else leave it as is.
            neuron.total_blocking_period_ +=
                std::numeric_limits<int64_t>::max() * ((neuron.total_blocking_period_ == 0) && was_negative);
        }
        else
        {
            neuron.total_blocking_period_ -= 1;
        }

        if (neuron.inhibitory_conductance_ < 1.0)
        {
            neuron.potential_ -=
                (neuron.potential_ - neuron.reversal_inhibitory_potential_) * neuron.inhibitory_conductance_;
        }
        else
        {
            neuron.potential_ = neuron.reversal_inhibitory_potential_;
        }

        if ((neuron.n_time_steps_since_last_firing_ > neuron.absolute_refractory_period_) &&
            (neuron.potential_ >= neuron.activation_threshold_ + neuron.dynamic_threshold_))
        {
            // Spike.
            neuron.dynamic_threshold_ += neuron.threshold_increment_;
            neuron.postsynaptic_trace_ += neuron.postsynaptic_trace_increment_;

            neuron.potential_ = neuron.potential_reset_value_;
            neuron.bursting_phase_ = neuron.bursting_period_;
            neuron.n_time_steps_since_last_firing_ = 0;
            spike = true;
        }

        if (neuron.potential_ < neuron.min_potential_)
        {
            neuron.potential_ = neuron.min_potential_;
        }

        if (spike)
        {
            neuron_indexes.push_back(index);
        }

        population.neurons_[index] = neuron;
    }

    if (!neuron_indexes.empty())
    {
        cuda::SpikeMessage res_message
        {
            .header_ = { .sender_uid_ = population.uid_, step_n},
            .neuron_indexes_ = neuron_indexes
        };

//        device_message_bus_.send_message(res_message);
        return res_message;
    }

    return {};
}


/*__device__ std::optional<core::messaging::SpikeMessage> CUDABackendImpl::calculate_population(
    CUDAPopulation<knp::neuron_traits::SynapticResourceSTDPBLIFATNeuron> &population,
    knp::backends::gpu::cuda::device_lib::CUDAVector<cuda::SynapticImpactMessage> &messages,
    std::uint64_t step_n)
{
//    SPDLOG_TRACE("Calculate resource-based STDP-compatible BLIFAT population {}.", std::string(population.get_uid()));
    return std::nullopt;
}*/


__device__ void CUDABackendImpl::calculate_projection(
    CUDAProjection<knp::synapse_traits::DeltaSynapse> &projection,
    knp::backends::gpu::cuda::device_lib::CUDAVector<cuda::SpikeMessage> &messages,
    std::uint64_t step_n)
{
    // Run:
    // knp::backends::cpu::calculate_delta_synapse_projection(
    //    projection, get_message_endpoint(), message_queue, get_step());


    // message_bus_.unload_messages<cuda::SpikeMessage>(projection.uid_, messages);

    // auto out_iter = calculate_delta_synapse_projection_data(projection, messages, future_messages, get_step());
    //
    // using SynapseType = typename ProjectionType::ProjectionSynapseType;
    // WeightUpdateSTDP<SynapseType>::init_projection(projection, messages, step_n);

    for (const knp::backends::gpu::cuda::SpikeMessage message : messages)
    {
        const auto &message_data = message.neuron_indexes_;
        for (size_t i = 0; i < message_data.size(); ++i)
        {
            const auto &spiked_neuron_index = message_data[i];

            for (size_t synapse_index = 0; synapse_index < projection.synapses_.size(); ++synapse_index)
            {
                CUDAProjection<knp::synapse_traits::DeltaSynapse>::Synapse synapse =
                    projection.synapses_[synapse_index];
                if (thrust::get<core::source_neuron_id>(synapse) != spiked_neuron_index) continue;
/*
                // WeightUpdateSTDP<SynapseType>::init_synapse(std::get<core::synapse_data>(synapse), step_n);
                const auto &synapse_params = thrust::get<core::synapse_data>(synapse);

                // The message is sent on step N - 1, received on step N.
                size_t future_step = synapse_params.delay_ + step_n - 1;
                knp::backends::gpu::cuda::SynapticImpact impact{
                    synapse_index, synapse_params.weight_, synapse_params.output_type_,
                    static_cast<uint32_t>(thrust::get<core::source_neuron_id>(synapse)),
                    static_cast<uint32_t>(thrust::get<core::target_neuron_id>(synapse))};
// TODO: use projection attribute.
                cuco::dynamic_map<std::uint64_t, cuda::SynapticImpactMessage> future_messages({}, {}, {});

                auto iter = future_messages.find(future_step);
                if (iter != future_messages.end())
                {
                    iter->second.impacts_.push_back(impact);
                }
                else
                {
                    cuda::SynapticImpactMessage message_out{
                        {projection.uid_, step_n},
                        projection.presynaptic_uid_,
                        projection.postsynaptic_uid_,
                        is_forcing<cuda::CUDAProjection<synapse_traits::DeltaSynapse>>(),
                        {impact}};
                    future_messages.insert(std::make_pair(future_step, message_out));
                }
*/
            }
        }
    }

/*
    // WeightUpdateSTDP<SynapseType>::modify_weights(projection);
    return future_messages.find(step_n);
    //

    if (out_iter != future_messages.end())
    {
        // Send a message and remove it from the queue.
        message_bus_.send_message(out_iter->second);
        future_messages.erase(out_iter);
    }
*/
}


__device__ void CUDABackendImpl::calculate_projection(
    CUDAProjection<knp::synapse_traits::AdditiveSTDPDeltaSynapse> &projection,
    knp::backends::gpu::cuda::device_lib::CUDAVector<cuda::SpikeMessage> &messages,
    std::uint64_t step_n)
{
    //SPDLOG_TRACE("Calculate AdditiveSTDPDelta synapse projection {}.", std::string(projection.get_uid()));
}


__device__ void CUDABackendImpl::calculate_projection(
    CUDAProjection<knp::synapse_traits::SynapticResourceSTDPDeltaSynapse> &projection,
    knp::backends::gpu::cuda::device_lib::CUDAVector<cuda::SpikeMessage> &messages,
    std::uint64_t step_n)
{
//    SPDLOG_TRACE("Calculate STDPSynapticResource synapse projection {}.", std::string(projection.get_uid()));
}


__host__ __device__ CUDABackendImpl::PopulationIterator CUDABackendImpl::begin_populations()
{
    return PopulationIterator{device_populations_.begin()};
}


__host__ __device__ CUDABackendImpl::PopulationConstIterator CUDABackendImpl::begin_populations() const
{
    return {device_populations_.cbegin()};
}


__host__ __device__ CUDABackendImpl::PopulationIterator CUDABackendImpl::end_populations()
{
    return PopulationIterator{device_populations_.end()};
}


__host__ __device__ CUDABackendImpl::PopulationConstIterator CUDABackendImpl::end_populations() const
{
    return device_populations_.cend();
}


__host__ __device__ CUDABackendImpl::ProjectionIterator CUDABackendImpl::begin_projections()
{
    return ProjectionIterator{device_projections_.begin()};
}


__host__ __device__ CUDABackendImpl::ProjectionConstIterator CUDABackendImpl::begin_projections() const
{
    return device_projections_.cbegin();
}


__host__ __device__ CUDABackendImpl::ProjectionIterator CUDABackendImpl::end_projections()
{
    return ProjectionIterator{device_projections_.end()};
}


__host__ __device__ CUDABackendImpl::ProjectionConstIterator CUDABackendImpl::end_projections() const
{
    return device_projections_.cend();
}

}  // namespace knp::backends::gpu::cuda
