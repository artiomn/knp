#include "hip/hip_runtime.h"
/**
 * @file message_bus.cu
 * @brief Message bus implementation.
 * @kaspersky_support Artiom N.
 * @date 21.02.2025
 * @license Apache 2.0
 * @copyright © 2024 AO Kaspersky Lab
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <boost/mp11/algorithm.hpp>
#include <cuda/std/detail/libcxx/include/algorithm>
#include <thrust/host_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <knp/meta/macro.h>
#include "message_bus.cuh"



namespace knp::backends::gpu::cuda
{
constexpr int threads_per_block = 256;


template <class T>
using DevVec = thrust::device_vector<T>;


    template <typename MessageType>
__device__ bool CUDAMessageBus::subscribe(const UID &receiver, const thrust::device_vector<UID> &senders)
{
    for (const auto &subscr : subscriptions_)
    {
        const bool is_sub_exists = ::cuda::std::visit(
            [&receiver](auto &arg)
            {
                using T = std::decay_t<decltype(arg)>;
                return std::is_same<MessageType, typename T::MessageType>::value &&
                       (arg.get_receiver_uid() == receiver);
            },
            subscr);

        // TODO: Check, that senders contain all senders in the formal parameter or `senders` has something new?
        if (is_sub_exists)
        {
            return false;
        }
    }

    subscriptions_.push_back(Subscription<MessageType>(receiver, senders));

    return true;
}


// Допустим мы хотим сделать плоский вектор для индексов сообщений, потому что иначе дофига раз выделять и пушить.
// Тогда нам надо иметь вектор размера "сумма всех отправителей для каждого получателя * число сообщений". Это значит 
// что? Это значит что нам надо будет передавать вектор на начало своего сегмента -- а потом разбирать результат.
// Чтобы разобрать результат, нужно иметь вектор индексов начала подписок. Это поможет потом разобрать сообщения... 
// А размер? Там должен быть размер


// Нам надо на хосте иметь размеры подписок, чтобы выделить вектор. Но подписки у нас в cuda-variant.
// Так что обрабатываем их в куда-ядре
__global__ void get_subscription_size(const CUDAMessageBus::SubscriptionContainer &subscriptions, 
                                      thrust::device_vector<uint64_t> &result)
{
    uint64_t id = threadIdx.x + blockIdx.x;
    if (id >= subscriptions.size()) return;
    uint64_t senders_num = ::cuda::std::visit([](const auto &s) 
    { 
        return s.get_senders().size(); 
    }, static_cast<SubscriptionVariant>(subscriptions[id]));
    result[id] = senders_num;
}


__host__ thrust::device_vector<uint64_t> get_senders_numbers(const CUDAMessageBus::SubscriptionContainer &subscriptions)
{
    thrust::device_vector<uint64_t> result(subscriptions.size());
    uint64_t num_threads = std::min<uint64_t>(threads_per_block, subscriptions.size());
    uint64_t num_blocks = (subscriptions.size() - 1) / threads_per_block + 1;
    get_subscription_size<<<num_blocks, num_threads>>>(subscriptions, result);
    return result;
}


// фигачим вектор, его задача в том чтобы для каждой подписки и каждого отправителя был свой вектор, зарезервированный под размер
// всех сообщений. На самом деле бы лучше сделать это для каждой подписки нужного типа, но логика будет сложнее.
DevVec<DevVec<DevVec<uint64_t>>> reserve_vector(const CUDAMessageBus::SubscriptionContainer &subscriptions, uint64_t size_z)
{
    DevVec<DevVec<DevVec<uint64_t>>> res;
    uint64_t size_x = subscriptions.size();
    thrust::host_vector<uint64_t> senders_numbers = get_senders_numbers(subscriptions);
    res.reserve(size_x);
    for (uint64_t i = 0; i < size_x; ++i)
    {
        DevVec<DevVec<uint64_t>> buf;
        buf.reserve(senders_numbers[i]);
        for (uint64_t j = 0; j < senders_numbers[i]; ++j)
        {
            DevVec<uint64_t> sub_buf;
            sub_buf.reserve(size_z);
            buf.push_back(std::move(sub_buf));
        }
        res.push_back(std::move(buf));
    }
    return res;
}


__device__ int find_by_sender(
    const thrust::device_vector<cuda::UID> &senders, 
    const CUDAMessageBus::MessageBuffer &messages,
    DevVec<DevVec<uint64_t>> &sub_message_indices,
    int type_index)
{
    int sender_index = blockIdx.x + threadIdx.x;
    if (sender_index >= senders.size()) return;
    cuda::UID uid = senders[sender_index];
    for (uint64_t i = 0; i < messages.size(); ++i) 
    {
        const cuda::MessageVariant &msg = messages[i];
        if (msg.index() != type_index) continue;
        cuda::UID msg_uid = ::cuda::std::visit([](const auto &msg) {return msg.header_.sender_uid_; }, msg);
        // if (msg_uid == uid) sub_message_indices[sender_index].push_back(msg_uid); 
        thrust::device_ptr<DevVec<uint64_t>> ptr = sub_message_indices.data();
        if (msg_uid == uid) (ptr + sender_index)->push_back(msg_uid); 

    }
}


// Так, надо найти вектор сообщений с известным получателем и известного типа. Это несложно.
// Что нам надо: для каждого получателя получить отправителей. Запустить поиск по отправителю. Собрать результаты в вектор.
// Что нам надо для верхней функции: набор подписок и индекс типа, вектор сообщений, размеры для набора и вектора.
// Ещё нужен вектор для результата: 
__global__ void find_messages_by_receiver(
        const CUDAMessageBus::SubscriptionContainer &subscriptions,
        const CUDAMessageBus::MessageBuffer &messages,
        DevVec<DevVec<DevVec<uint64_t>>> &message_indices,
        int type_index)
{
    uint64_t sub_index = threadIdx.x + blockIdx.x;
    if (sub_index >= subscriptions.size()) return;
    const SubscriptionVariant &subscription = subscriptions[sub_index];
    if (subscription.index() != type_index) return;
    const DevVec<cuda::UID> &senders = ::cuda::std::visit([](auto &sub) { return sub.get_senders(); }, subscription);
    uint64_t buf_size = messages.size();

    // Find number of threads and blocks
    const int num_threads = std::min<int>(threads_per_block, subscriptions.size());
    const int num_blocks = subscriptions.size() / threads_per_block + 1;
    find_by_sender<<<num_blocks, num_threads>>>(senders, messages, message_indices[sub_index], type_index);
}


template<class MessageType>
__host__ thrust::device_vector<thrust::device_vector<thrust::device_vector<uint64_t>>> CUDAMessageBus::index_messages()
{
    uint64_t buf_size = messages_to_route_.size();
    // constexpr int type_index = boost::mp_find<MessageVariant, MessageType>::value;
    constexpr int type_index = 0; // TODO fix the code above.
    //Reserve memory:
    // Triple vector: receiver * senders * all_messages
    auto found_messages_indices = reserve_vector(subscriptions_, messages_to_route_.size());
    // Find number of threads and blocks and run the core.
    const int num_threads = std::min<int>(threads_per_block, subscriptions_.size());
    const int num_blocks = subscriptions_.size() / threads_per_block + 1;
    find_messages_by_receiver<<<num_blocks, num_threads>>>(subscriptions_, messages_to_route_, found_messages_indices, type_index);
    return found_messages_indices;
}




template <typename MessageType>
__device__ bool CUDAMessageBus::unsubscribe(const UID &receiver)
{
    auto sub_iter = thrust::find_if(thrust::device, subscriptions_.begin(), subscriptions_.end(),
    [&receiver](const cuda::SubscriptionVariant &subscr) -> bool
    {
        return std::visit([&receiver](const auto &arg)
        {
            using T = std::decay_t<decltype(arg)>;
            return std::is_same<MessageType, typename T::MessageType>::value && (arg.get_receiver_uid() == receiver);
        }, subscr);
    });

    if (subscriptions_.end() == sub_iter) return false;

    subscriptions_.erase(sub_iter);

    return true;
}


__device__ void CUDAMessageBus::remove_receiver(const UID &receiver)
{
    for (auto sub_iter = subscriptions_.begin(); sub_iter != subscriptions_.end(); ++sub_iter)
    {
/*        ::cuda::std::visit([&receiver](auto &&arg)
        {
            return arg.get_receiver_uid() == receiver;
        }, *sub_iter);
*/
    }

/*    if (subscriptions_.end() == sub_iter) return;

    subscriptions_.erase(sub_iter);*/
}


// This is not threadsafe, make sure it's not run in parallel.
__device__ void CUDAMessageBus::send_message(const cuda::MessageVariant &message)
{
    messages_to_route_.push_back(message);
}


__device__ size_t CUDAMessageBus::step()
{
    // Как у нас работает бэк: каждая популяция получает входные сообщения и формирует, но не отправляет выходное сообщение. 
    // когда все сообщения получены, мы чистим шину, получаем сообщения от популяций (в цикле) и отправляем их в эндпойнт.
    // потом мы проходим по проекциям (параллельно или нет), и они формируют сообщения
    // мы чистим шину от спайковых сообщений, получаем сообщения от проекций (в цикле) и отправляем их в эндпойнт.
    // таким образом, конкретного step-а у нас попросту не образуется. Степ состоит из clear(), 
    // for(...) if(get_num_messages > 0) send_message(get_stored_messages) и sync(). 
    // Все эти функции вызываются из бэкенда чем-то вроде do_message_exchange().
    return 1;
}


__device__ size_t CUDAMessageBus::route_messages()
{
    size_t count = 0;
    size_t num_messages = step();

    while (num_messages != 0)
    {
        count += num_messages;
        num_messages = step();
    }

    return count;
}


template <class MessageType>
__device__ void CUDAMessageBus::receive_messages(const cuda::UID &receiver_uid,
        thrust::device_vector<MessageType> &result_messages)
{
    // locate messages
    
}


}  // namespace knp::backends::gpu::cuda
