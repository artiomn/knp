#include "hip/hip_runtime.h"
/**
 * @file cuda_test.cu
 * @brief CUDA backend test.
 * @kaspersky_support Artiom N.
 * @date 26.02.2025
 * @license Apache 2.0
 * @copyright © 2025 AO Kaspersky Lab
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// #include <knp/backends/gpu-cuda/backend.h>
#include <knp/core/message_bus.h>
#include <knp/core/population.h>
#include <knp/core/projection.h>

#include <generators.h>
#include <spdlog/spdlog.h>
#include <tests_common.h>

#include <functional>
#include <iostream>
#include <vector>


#include "../../backends/gpu/cuda-backend/impl/cuda_lib/vector.cuh"
#include "../../backends/gpu/cuda-backend/impl/cuda_bus/message_bus.cuh"
#include "../../backends/gpu/cuda-backend/impl/cuda_bus/messaging.cuh"
#include "../../backends/gpu/cuda-backend/impl/uid.cuh"


// using Population = knp::backends::gpu::CUDABackend::PopulationVariants;
// using Projection = knp::backends::gpu::CUDABackend::ProjectionVariants;


namespace knp::testing
{


// struct MessageBusTandem
// {
//     MessageBusTandem() : cpu_(knp::core::MessageBus::construct_bus()), gpu_(cpu_.create_endpoint())
//     {}
//     knp::core::MessageBus cpu_;
//     knp::backends::gpu::cuda::CUDAMessageBus gpu_;
// };


template <class Endpoint>
bool send_messages_smallest_network(const knp::core::UID &in_channel_uid, Endpoint &endpoint, knp::core::Step step)
{
    if (step % 5 == 0)
    {
        knp::core::messaging::SpikeMessage message{{in_channel_uid, 0}, {0}};
        endpoint.send_message(message);
        return true;
    }
    return false;
}


template <class Endpoint>
bool receive_messages_smallest_network(const knp::core::UID &out_channel_uid, Endpoint &endpoint)
{
    endpoint.receive_all_messages();
    // Write the steps on which the network sends a spike.
    if (!endpoint.template unload_messages<knp::core::messaging::SpikeMessage>(out_channel_uid).empty()) return true;
    return false;
}


__device__ void prn()
{
    printf("Hello from GPU! Thread ID: %d\n", threadIdx.x);

    knp::backends::gpu::cuda::device_lib::CudaVector<int> cv;

    cv.reserve(2);

    printf("cv.size() = %lu\n", cv.size());
    cv.push_back(1);
    printf("cv.size() = %lu, v = %d\n", cv.size(), cv[0]);
    auto v = cv.pop_back();
    printf("p1 = %d\n", v);
    printf("cv.size() = %lu\n", cv.size());
    cv.resize(10);

    for (int i = 0; i < cv.size(); ++i) printf("i0 = %d\n", cv[i]);
    for (int i = 0; i < cv.size(); ++i) cv.set(i, i);
    for (int i = 0; i < cv.size(); ++i) printf("i1 = %d\n", cv[i]);
    cv.resize(5);
    for (int i = 0; i < cv.size(); ++i) printf("i2 = %d\n", cv[i]);
    cv.reserve(15);
    for (int i = 0; i < cv.size(); ++i) printf("i3 = %d\n", cv[i]);
}


__global__ void run_bus()
{
    prn();
}


/*TEST(CudaBackendSuite, CUDADevice)  // cppcheck-suppress syntaxError
{
    auto gpus = knp::devices::gpu::list_cuda_processors();
    for (const auto &gpu : gpus)
    {
        auto gpu_ptr = dynamic_cast<const knp::devices::gpu::CUDA *>(&gpu);
        SPDLOG_INFO(
            "GPU name: {}, warp size = {}, power = {}", gpu.get_name(), gpu_ptr->get_warp_size(), gpu.get_power());
    }
}*/

TEST(CudaContainerSuite, Memcpy)
{
    const uint64_t val = 112;
    uint64_t *val_gpu;
    uint64_t val_cpu = 0;
    hipMalloc(&val_gpu, sizeof(uint64_t));
    hipMemcpy(val_gpu, &val, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(&val_cpu, val_gpu, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(val_gpu);
    ASSERT_EQ(val, val_cpu);
}


TEST(CudaContainerSuite, MemcpyArray)
{
    const cuda::std::array<uint64_t, 4> array{1, 2, 3, 4};
    cuda::std::array<uint64_t, 4> *array_gpu;
    cuda::std::array<uint64_t, 4> array_cpu{4, 3, 2, 1};
    hipMalloc(&array_gpu, sizeof(cuda::std::array<uint64_t, 4>));
    hipMemcpy(array_gpu, &array, sizeof(array), hipMemcpyHostToDevice);
    hipMemcpy(&array_cpu, array_gpu, sizeof(array), hipMemcpyDeviceToHost);
    hipFree(array_gpu);
    ASSERT_EQ(array, array_cpu);
}


TEST(CudaContainerSuite, CopyKernel)
{
    namespace knp_cuda = knp::backends::gpu::cuda;
    uint64_t *array_from = nullptr;
    uint64_t *array_to = nullptr;
    hipMalloc(&array_from, 8 * sizeof(uint64_t));
    hipMalloc(&array_to, 8 * sizeof(uint64_t));

    std::vector<uint64_t> vec_from = {3, 2, 4, 5, 1, 0, 4, 0};
    std::vector<uint64_t> vec_out(vec_from.size());
    hipMemcpy(array_from, vec_from.data(), 8 * sizeof(uint64_t), hipMemcpyHostToDevice);
    knp_cuda::device_lib::copy_kernel<<<1, 8>>>(0, 8, array_to, array_from);
    hipMemcpy(vec_out.data(), array_to, vec_from.size() * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(array_from);
    hipFree(array_to);
    ASSERT_EQ(vec_from, vec_out);
}


TEST(CudaContainerSuite, EqualKernel)
{
    // TODO: RAII !!!
    namespace knp_cuda = knp::backends::gpu::cuda;
    typedef uint64_t int_type;
    constexpr int num_values = 8;
    int_type *array = nullptr;
    int_type *array_same = nullptr;
    int_type *array_other = nullptr;
    hipMalloc(&array, num_values * sizeof(int_type));
    hipMalloc(&array_same, num_values * sizeof(int_type));
    hipMalloc(&array_other, num_values * sizeof(int_type));

    std::vector<int_type> values = {1, 2, 1, 12, 9, 9, 3, 5};
    std::vector<int_type> other_values = {1, 2, 3, 4, 5, 6, 7, 8};

    uint64_t mem_size = num_values * sizeof(int_type);
    hipMemcpy(array, values.data(), mem_size, hipMemcpyHostToDevice);
    hipMemcpy(array_same, values.data(), mem_size, hipMemcpyHostToDevice);
    hipMemcpy(array_other, other_values.data(), mem_size, hipMemcpyHostToDevice);

    bool result = false;
    bool *gpu_result;

    hipMalloc(&gpu_result, sizeof(bool));
    knp_cuda::device_lib::equal_kernel<<<1, 1>>>(array, array_same, num_values, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_TRUE(result);
    knp_cuda::device_lib::equal_kernel<<<1, 1>>>(array, array_other, num_values, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_FALSE(result);
    hipFree(array);
    hipFree(array_same);
    hipFree(array_other);
    hipFree(gpu_result);
}


TEST(CudaContainerSuite, VectorPushBack)
{
    namespace knp_cuda = knp::backends::gpu::cuda;
    knp_cuda::device_lib::CudaVector<uint64_t> cuda_vec;

    ASSERT_EQ(cuda_vec.size(), 0);
    std::cout << cuda_vec << std::endl;
    cuda_vec.push_back(1);
    std::cout << cuda_vec << std::endl;
    cuda_vec.push_back(2);
    std::cout << cuda_vec << std::endl;
    cuda_vec.push_back(3);
    std::cout << cuda_vec << std::endl;

    ASSERT_EQ(cuda_vec.size(), 3);
    ASSERT_GE(cuda_vec.capacity(), 3);
    std::vector<uint64_t> exp_results{1, 2, 3};
    knp_cuda::device_lib::CudaVector res(exp_results.data(), exp_results.size());
    // ASSERT_EQ(cuda_vec, exp_results);
    ASSERT_EQ(cuda_vec[0], 1);
    ASSERT_EQ(cuda_vec[1], 2);
    ASSERT_EQ(cuda_vec[2], 3);
    ASSERT_EQ(cuda_vec, res);
}


TEST(CudaBaseSuite, CudaVectorConstruct)
{
    namespace knp_cuda = knp::backends::gpu::cuda;

    knp_cuda::device_lib::CudaVector<uint64_t> cuda_vec_1;
    knp_cuda::device_lib::CudaVector<uint64_t> cuda_vec_2(10);

    ASSERT_EQ(cuda_vec_1.size(), 0);
    ASSERT_EQ(cuda_vec_2.size(), 10);
}


TEST(CudaBackendSuite, CudaUidConversionTest)
{
    knp::core::UID orig_uid;
    knp::backends::gpu::cuda::UID cuda_uid = knp::backends::gpu::cuda::to_gpu_uid(orig_uid);
    knp::core::UID restored_uid = knp::backends::gpu::cuda::to_cpu_uid(cuda_uid);
    ASSERT_EQ(orig_uid, restored_uid);
}


TEST(CudaBackendSuite, MessagesTest)
{
    namespace knp_cuda = knp::backends::gpu::cuda;

    knp_cuda::SpikeMessage message_1;
    knp_cuda::SynapticImpactMessage message_2;
    ASSERT_EQ(message_1.neuron_indexes_.size(), 0);
    ASSERT_EQ(message_2.impacts_.size(), 0);
}


TEST(CudaBackendSuite, CudaHostSubscription)
{
    namespace knp_cuda = knp::backends::gpu::cuda;

    knp_cuda::UID receiver_uid = knp_cuda::to_gpu_uid(knp::core::UID{});
    knp_cuda::UID sender_1 = knp_cuda::to_gpu_uid(knp::core::UID{}), sender_2 = knp_cuda::to_gpu_uid(knp::core::UID{});
    ASSERT_NE(sender_1, sender_2);
    knp_cuda::Subscription<knp_cuda::SpikeMessage> subscription(receiver_uid, {sender_1});

    ASSERT_EQ(subscription.get_senders().size(), 1);
    ASSERT_TRUE(subscription.has_sender(sender_1));
    ASSERT_FALSE(subscription.has_sender(sender_2));
}


TEST(CudaBackendSuite, CudaBusSubscription)
{
    // using knp::backends::gpu::cuda::to_gpu_uid;
    // using knp::backends::gpu::cuda::device_lib::CudaVector;
    // using knp::backends::gpu::cuda::UID;
    // MessageBusTandem bus_pair;
    // knp::core::UID sender_1, sender_2, receiver_1, receiver_2;
    // CudaVector<UID> senders_1, senders_2;
    // senders_1.push_back(to_gpu_uid(sender_1));
    // senders_1.push_back(to_gpu_uid(sender_2));
    // bus_pair.gpu_.subscribe<knp::backends::gpu::cuda::SpikeMessage>(
    //         to_gpu_uid(receiver_1), senders_1);
    // senders_2.push_back(to_gpu_uid(sender_1));
    // bus_pair.gpu_.subscribe<knp::backends::gpu::cuda::SpikeMessage>(
    //     to_gpu_uid(receiver_2), senders_2);
    // ASSERT_EQ(bus_pair.gpu_.get_subscriptions().size(), 2);


    // const knp::backends::gpu::cuda::SubscriptionVariant &sub_v = bus_pair.gpu_.get_subscriptions()[0];
    // const auto &sub = ::cuda::std::get<knp::backends::gpu::cuda::Subscription<
    //                     knp::backends::gpu::cuda::SpikeMessage>>(sub_v);
    // ASSERT_EQ(sub.get_senders().size(), 2);
}


TEST(CudaBackendSuite, SmallestNetwork)
{
    // Create a single-neuron neural network: input -> input_projection -> population <=> loop_projection.

    namespace kt = knp::testing;

    // Spikes on steps "5n + 1" (input) and on "previous_spike_n + 6" (positive feedback loop).
    // const std::vector<knp::core::Step> expected_results = {1, 6, 7, 11, 12, 13, 16, 17, 18, 19};
    // ASSERT_EQ(results, expected_results);
}


TEST(CudaBackendSuite, NeuronsGettingTest)
{
    // const knp::testing::MTestingBack backend;

    // auto s_neurons = backend.get_supported_neurons();

    // ASSERT_LE(s_neurons.size(), boost::mp11::mp_size<knp::neuron_traits::AllNeurons>());
    // ASSERT_EQ(s_neurons[0], "BLIFATNeuron");
}


TEST(CudaBackendSuite, SynapsesGettingTest)
{
    // const knp::testing::MTestingBack backend;

    // auto s_synapses = backend.get_supported_synapses();

    // ASSERT_LE(s_synapses.size(), boost::mp11::mp_size<knp::synapse_traits::AllSynapses>());
    // ASSERT_EQ(s_synapses[0], "DeltaSynapse");
}

}  // namespace knp::testing
