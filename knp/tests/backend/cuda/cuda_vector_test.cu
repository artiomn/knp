#include "hip/hip_runtime.h"
/**
 * @file cuda_test.cu
 * @brief CUDA backend test.
 * @kaspersky_support Artiom N.
 * @date 26.02.2025
 * @license Apache 2.0
 * @copyright © 2025 AO Kaspersky Lab
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// #include <knp/backends/gpu-cuda/backend.h>
#include <knp/core/message_bus.h>
#include <knp/core/population.h>
#include <knp/core/projection.h>

#include <generators.h>
#include <spdlog/spdlog.h>
#include <tests_common.h>

#include <functional>
#include <iostream>
#include <vector>


#include "../../../backends/gpu/cuda-backend/impl/cuda_lib/vector.cuh"
#include "../../../backends/gpu/cuda-backend/impl/cuda_lib/vector_kernels.cuh"
#include "../../../backends/gpu/cuda-backend/impl/uid.cuh"


// using Population = knp::backends::gpu::CUDABackend::PopulationVariants;
// using Projection = knp::backends::gpu::CUDABackend::ProjectionVariants;


namespace knp::testing
{

TEST(CudaVectorSuite, Memcpy)
{
    hipDeviceReset();
    const uint64_t val = 112;
    uint64_t *val_gpu;
    uint64_t val_cpu = 0;
    hipMalloc(&val_gpu, sizeof(uint64_t));
    hipMemcpy(val_gpu, &val, sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(&val_cpu, val_gpu, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(val_gpu);
    ASSERT_EQ(val, val_cpu);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
}


TEST(CudaVectorSuite, MemcpyArray)
{
    hipDeviceReset();
    const cuda::std::array<uint64_t, 4> array{1, 2, 3, 4};
    cuda::std::array<uint64_t, 4> *array_gpu;
    cuda::std::array<uint64_t, 4> array_cpu{4, 3, 2, 1};
    hipMalloc(&array_gpu, sizeof(cuda::std::array<uint64_t, 4>));
    hipMemcpy(array_gpu, &array, sizeof(array), hipMemcpyHostToDevice);
    hipMemcpy(&array_cpu, array_gpu, sizeof(array), hipMemcpyDeviceToHost);
    hipFree(array_gpu);
    ASSERT_EQ(array, array_cpu);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
}


TEST(CudaVectorSuite, CopyKernel)
{
    hipDeviceReset();
    namespace knp_cuda = knp::backends::gpu::cuda;
    uint64_t *array_from = nullptr;
    uint64_t *array_to = nullptr;
    hipMalloc(&array_from, 8 * sizeof(uint64_t));
    hipMalloc(&array_to, 8 * sizeof(uint64_t));

    std::vector<uint64_t> vec_from = {3, 2, 4, 5, 1, 0, 4, 0};
    std::vector<uint64_t> vec_out(vec_from.size());
    hipMemcpy(array_from, vec_from.data(), 8 * sizeof(uint64_t), hipMemcpyHostToDevice);
    knp_cuda::device_lib::copy_kernel<<<1, 8>>>(array_from, 0, 8, array_to);
    hipMemcpy(vec_out.data(), array_to, vec_from.size() * sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipFree(array_from);
    hipFree(array_to);
    ASSERT_EQ(vec_from, vec_out);
    ASSERT_EQ(hipGetLastError(), hipSuccess);

}


__global__ void copy_uid_kernel(size_t begin, size_t end, knp::backends::gpu::cuda::UID* data_to,
                                const knp::backends::gpu::cuda::UID* data_from)
{
    printf("Copy uid kernel, begin: %lu, end: %lu, sizeof data %lu\n", begin, end,
           sizeof(knp::backends::gpu::cuda::UID));
    if (end <= begin) return;
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Copy kernel: index %lu, from %p to %p\n", i, data_from + begin + i, data_to + begin + i);
    if (i >= end - begin) return;
    new (data_to + begin + i) knp::backends::gpu::cuda::UID(*(data_from + begin + i));
    // *(data_to + begin + i) = *(data_from + begin + i);
}

TEST(CudaVectoSuite, CopyUidKernel)
{
    namespace knp_cuda = knp::backends::gpu::cuda;
    knp_cuda::UID *array_from = nullptr;
    knp_cuda::UID *array_to = nullptr;
    auto error = hipGetLastError();
    if (error != hipSuccess)
        std::cout << "ERROR BEFORE RESET: " << hipGetErrorString(error) << std::endl;
    hipDeviceReset();
    hipMalloc(&array_from, 4 * sizeof(knp_cuda::UID));
    hipMalloc(&array_to, 4 * sizeof(knp_cuda::UID));
    error = hipGetLastError();
    if (error != hipSuccess)
        std::cout << "ERROR 0: " << hipGetErrorString(error) << std::endl;
    knp_cuda::UID uid1 = knp_cuda::to_gpu_uid(knp::core::UID{}), uid2 = knp_cuda::to_gpu_uid(knp::core::UID{});
    knp_cuda::UID uid3 = knp_cuda::to_gpu_uid(knp::core::UID{}), uid4 = knp_cuda::to_gpu_uid(knp::core::UID{});
    std::vector<knp_cuda::UID> vec_from{uid1, uid2, uid3, uid4};
    std::vector<knp_cuda::UID> vec_out(vec_from.size());
    error = hipGetLastError();
    hipMemcpy(array_from, vec_from.data(), 4 * sizeof(knp_cuda::UID), hipMemcpyHostToDevice);

    if (error != hipSuccess)
        std::cout << "ERROR 1: " << hipGetErrorString(error) << std::endl;

    // copy_uid_kernel<<<1, 4>>>(0, 4, array_to, array_from);
    knp_cuda::device_lib::copy_kernel<<<1, 4>>>(array_from, 0, 4, array_to);
    error = hipGetLastError();
    if (error != hipSuccess)
        std::cout << "ERROR 2: " << hipGetErrorString(error) << std::endl;
    hipMemcpy(vec_out.data(), array_to, vec_from.size() * sizeof(knp_cuda::UID), hipMemcpyDeviceToHost);
    hipFree(array_from);
    hipFree(array_to);
    ASSERT_EQ(vec_from, vec_out);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
}


TEST(CudaVectorSuite, EqualKernel)
{
    hipDeviceReset();
    // TODO: RAII !!!
    namespace knp_cuda = knp::backends::gpu::cuda;
    typedef uint64_t int_type;
    constexpr int num_values = 8;
    int_type *array = nullptr;
    int_type *array_same = nullptr;
    int_type *array_other = nullptr;
    hipMalloc(&array, num_values * sizeof(int_type));
    hipMalloc(&array_same, num_values * sizeof(int_type));
    hipMalloc(&array_other, num_values * sizeof(int_type));

    std::vector<int_type> values = {1, 2, 1, 12, 9, 9, 3, 5};
    std::vector<int_type> other_values = {1, 2, 3, 4, 5, 6, 7, 8};

    uint64_t mem_size = num_values * sizeof(int_type);
    hipMemcpy(array, values.data(), mem_size, hipMemcpyHostToDevice);
    hipMemcpy(array_same, values.data(), mem_size, hipMemcpyHostToDevice);
    hipMemcpy(array_other, other_values.data(), mem_size, hipMemcpyHostToDevice);

    bool result = false;
    bool *gpu_result;

    hipMalloc(&gpu_result, sizeof(bool));
    knp_cuda::device_lib::equal_kernel<<<1, 1>>>(array, array_same, num_values, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_TRUE(result);
    knp_cuda::device_lib::equal_kernel<<<1, 1>>>(array, array_other, num_values, gpu_result);
    hipMemcpy(&result, gpu_result, sizeof(bool), hipMemcpyDeviceToHost);
    ASSERT_FALSE(result);
    hipFree(array);
    hipFree(array_same);
    hipFree(array_other);
    hipFree(gpu_result);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
}


TEST(CudaVectorSuite, VectorPushBack)
{
    namespace knp_cuda = knp::backends::gpu::cuda;
    knp_cuda::device_lib::CUDAVector<uint64_t> cuda_vec;

    ASSERT_EQ(hipGetLastError(), hipSuccess);
    ASSERT_EQ(cuda_vec.size(), 0);
    std::cout << cuda_vec << std::endl;
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    cuda_vec.push_back(1);
    std::cout << cuda_vec << std::endl;
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    cuda_vec.push_back(2);
    std::cout << cuda_vec << std::endl;
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    cuda_vec.push_back(3);
    std::cout << cuda_vec << std::endl;
    ASSERT_EQ(hipGetLastError(), hipSuccess);
    ASSERT_EQ(cuda_vec.size(), 3);
    ASSERT_GE(cuda_vec.capacity(), 3);
    std::vector<uint64_t> exp_results{1, 2, 3};
    knp_cuda::device_lib::CUDAVector res(exp_results.data(), exp_results.size());
    // ASSERT_EQ(cuda_vec, exp_results);
    ASSERT_EQ(cuda_vec[0], 1);

    ASSERT_EQ(cuda_vec[1], 2);
    ASSERT_EQ(cuda_vec[2], 3);
    ASSERT_EQ(cuda_vec, res);
    ASSERT_EQ(hipGetLastError(), hipSuccess);

}


TEST(CudaVectorSuite, CUDAVectorConstruct)
{
    namespace knp_cuda = knp::backends::gpu::cuda;

    knp_cuda::device_lib::CUDAVector<uint64_t> cuda_vec_1;
    knp_cuda::device_lib::CUDAVector<uint64_t> cuda_vec_2(10);

    ASSERT_EQ(cuda_vec_1.size(), 0);
    ASSERT_EQ(cuda_vec_2.size(), 10);
    ASSERT_EQ(hipGetLastError(), hipSuccess);
}

}  // namespace knp::testing
